#include "../include/dev_dat_t.hpp"

void testing_devdat() {
    int driver_version , runtime_version;
    hipDriverGetVersion( &driver_version );
    hipRuntimeGetVersion ( &runtime_version );
    std::cout << driver_version << "\t" << runtime_version << std::endl;

    // generate 32M random numbers serially
    thrust::host_vector<cudaT> h_vec(2 << 3, 0);
    // std::generate(h_vec.begin(), h_vec.end(), rand);
    h_vec[4] = 2.0;
    h_vec[10] = -1.0;

    print_range("Host vector", h_vec.begin(), h_vec.end());
    
    // transfer data to the device
    dev_vec d_vec = h_vec;
    print_range("Device vector", d_vec.begin(), d_vec.end());

    odesolver::DevDatC sampled_coordinates(d_vec, 2);

    sampled_coordinates.print_dim_by_dim();
    sampled_coordinates.print_elem_by_elem();

    // Testing the copy constructor
    std::cout << "Testing the copy constructor" << std::endl;
    odesolver::DevDatC a = sampled_coordinates;
    std::cout << "a.size(): " << a.size() << "; sampled_coordinates.size(): " << sampled_coordinates.size() << std::endl;
    a.print_dim_by_dim();
    a.print_elem_by_elem();

    // Testing the assignment operator (copy-and-swap-idiom)
    std::cout << "Testing the assignment operator (copy + assign)" << std::endl;
    odesolver::DevDatC b;
    b = sampled_coordinates;
    std::cout << "b.size(): " << b.size() << "; sampled_coordinates.size(): " << sampled_coordinates.size() << std::endl;
    b.print_dim_by_dim();
    b.print_elem_by_elem();

    // Testing the move operator
    std::cout << "Testing the move operator" << std::endl;
    odesolver::DevDatC c = std::move(sampled_coordinates);
    std::cout << "c.size(): " << c.size() << "; sampled_coordinates.size(): " << sampled_coordinates.size() << std::endl;
    c.print_dim_by_dim();
    c.print_elem_by_elem();

    // Transpose data
    auto transposed_vec_vec = c.transpose_device_data();
    odesolver::DevDatC tranposed_dev_dat(transposed_vec_vec);
    tranposed_dev_dat.print_dim_by_dim();
    tranposed_dev_dat.print_elem_by_elem();

    // Fill vector
    odesolver::DevDatC d(2, 8); // dim, N
    d.print_dim_by_dim();
    d.print_elem_by_elem();
    d.fill_by_vec(d_vec);
    d.print_dim_by_dim();
    d.print_elem_by_elem();

    // ...
    std::cout << "d.dim_size(): " << d.dim_size() << "; d.n_elems(): " << d.n_elems() << std::endl;
}