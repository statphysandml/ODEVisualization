#include "hip/hip_runtime.h"
#include "../include/flow_equations_t.hpp"

odesolver::DevDatC get_fixed_points()
{
    // double sigma = 10.0;
    double beta = 8.0/3.0;
    double rho = 28.0;
    odesolver::DevDatC fixed_points(std::vector<std::vector<double>>{
        {0.0, 0.0, 0.0},
        {std::sqrt(beta * (rho - 1)), std::sqrt(beta * (rho - 1)), rho - 1},
        {-std::sqrt(beta * (rho - 1)), -std::sqrt(beta * (rho - 1)), rho - 1}
    });
    return fixed_points;
}

void compute_vertex_velocities_t()
{
    auto flow_equations_ptr = generate_flow_equations<LorentzAttractorFlowEquations>(0);
    auto fixed_points = get_fixed_points();
    fixed_points.print_dim_by_dim();

    auto vertex_velocities = compute_vertex_velocities(fixed_points, flow_equations_ptr.get());

    vertex_velocities.print_dim_by_dim();
}

odesolver::DevDatC compute_jacobians()
{

    auto jacobian_equations_ptr = generate_jacobian_equations<LorentzAttractorJacobianEquations>(0);

    auto fixed_points = get_fixed_points();
    fixed_points.print_elem_by_elem();

    auto jacobian_elements = compute_jacobian_elements(fixed_points, jacobian_equations_ptr.get());

    jacobian_elements.print_elem_by_elem();

    return std::move(jacobian_elements);
}

std::vector<std::vector<double>> compute_jacobian_elements_t()
{
    auto jacobian_elements = compute_jacobians();
    return jacobian_elements.transpose_device_data();
}