#include "hip/hip_runtime.h"
#include <odesolver/flow_equations/jacobian_equation.hpp>


namespace odesolver {
    namespace flowequations {
        odesolver::DevDatC compute_jacobian_elements(const odesolver::DevDatC &coordinates, JacobianEquationWrapper * jacobian_equations)
        {
            const uint dim = coordinates.dim_size();
            auto number_of_coordinates = coordinates.n_elems();
            odesolver::DevDatC jacobian_elements(pow(dim, 2), number_of_coordinates);
            // Evaluate jacobian for each element separately
            for(auto matrix_idx = 0; matrix_idx < pow(dim, 2); matrix_idx ++) {
                (*jacobian_equations)(jacobian_elements[matrix_idx], coordinates, matrix_idx);
            }
            return std::move(jacobian_elements);
        }


        void compute_jacobian_elements(const odesolver::DevDatC &coordinates, odesolver::DevDatC &jacobian_elements, JacobianEquationWrapper * jacobian_equations)
        {
            // Evaluate jacobian for each element separately
            for(auto matrix_idx = 0; matrix_idx < pow(coordinates.dim_size(), 2); matrix_idx ++) {
                (*jacobian_equations)(jacobian_elements[matrix_idx], coordinates, matrix_idx);
            }
        }
    }
}
