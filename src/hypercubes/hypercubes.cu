#include "hip/hip_runtime.h"
#include "../../include/hypercubes/hypercubes.hpp"



struct greater_than_zero
{
    template< typename T>
    __host__ __device__
    T operator()(const T &val) const
    {
        return val > 0;
    }
};


struct accumulate_n_branches
{
    accumulate_n_branches(const int dim_index_, int init_) : dim_index(dim_index_), sum(init_)
    {}

    __host__
    int operator()(const std::vector<int> elem)
    {
        sum = sum * elem[dim_index];
        return sum;
    }

    int sum;
    const int dim_index;
};


struct compute_axis_index_of_coor
{
    compute_axis_index_of_coor(const cudaT lambda_offset_, const cudaT delta_lambda_, const int n_branch_per_depth_at_dim_) :
        lambda_offset(lambda_offset_), delta_lambda(delta_lambda_), n_branch_per_depth_at_dim(n_branch_per_depth_at_dim_)
    {}

    __host__ __device__
    int operator()(const cudaT &coordinate)
    {
        return (int((coordinate+lambda_offset)/delta_lambda) % n_branch_per_depth_at_dim);
    }

    const cudaT lambda_offset;
    const cudaT delta_lambda;
    const int n_branch_per_depth_at_dim;
};


struct sum_axis_indices_to_cube_index
{
    sum_axis_indices_to_cube_index(const int accum_n_branch_per_depth_) :
        accum_n_branch_per_depth(accum_n_branch_per_depth_)
    {}

    __host__ __device__
    int operator()(const int &axis_index, const int &cube_index)
    {
        return cube_index + axis_index * accum_n_branch_per_depth;
    }

    const int accum_n_branch_per_depth;
};


/* Remark: Generating an array for accum_n_branches_per_dim in advance as it is done for the other expanded vectors
 * makes no sense since the values do not differ from node to node (only the depth values) (reconsider this argument)
 * -> this is probably not true, only valid argument would be redundancy */

struct compute_depth_vertex_coor_weight
{
    compute_depth_vertex_coor_weight(const int n_branch_per_depth_, const int accum_n_branch_per_depth_,
            const int depth_weight_divisor_, dev_vec_int const& accum_n_branches_per_dim_) :
            n_branch_per_depth(n_branch_per_depth_), accum_n_branch_per_depth(accum_n_branch_per_depth_),
            depth_weight_divisor(depth_weight_divisor_),
            accum_n_branches_per_dim_ptr(thrust::raw_pointer_cast(&accum_n_branches_per_dim_[0]))
    {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        int cube_index = thrust::get<0>(t);
        int maximum_cube_depth = thrust::get<1>(t) + 1; // = k
        int current_val = thrust::get<2>(t);

        thrust::get<2>(t) += ((cube_index/ accum_n_branch_per_depth) % n_branch_per_depth) * accum_n_branches_per_dim_ptr[maximum_cube_depth] / depth_weight_divisor;
    }

    const int n_branch_per_depth;
    const int accum_n_branch_per_depth;
    const int depth_weight_divisor;
    const int * accum_n_branches_per_dim_ptr;
};


struct compute_inner_vertex_coor
{
    compute_inner_vertex_coor(const int dim_index_) : dim_index(dim_index_)
    {}

    __host__ __device__
    int operator()(const int &ith_inner_index) const
    {
        return int(ith_inner_index/ pow(2, dim_index)) % 2;
    }

    const int dim_index;
};


struct finalize_vertex_computation
{
    finalize_vertex_computation(const cudaT lambda_range_, const cudaT lambda_offset_,
            dev_vec_int const& accum_n_branches_per_dim_) :
            lambda_range(lambda_range_), lambda_offset(lambda_offset_),
            accum_n_branches_per_dim_ptr(thrust::raw_pointer_cast(&accum_n_branches_per_dim_[0]))
    {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        int reference_vertex = thrust::get<0>(t);
        int maximum_cube_depth  = thrust::get<1>(t) + 1; // = k
        int inner_vertex_coor = thrust::get<2>(t);

        thrust::get<3>(t) = (reference_vertex + inner_vertex_coor) * lambda_range / accum_n_branches_per_dim_ptr[maximum_cube_depth] + lambda_offset;
    }

    const cudaT lambda_range;
    const cudaT lambda_offset;
    const int * accum_n_branches_per_dim_ptr;
};


struct finalize_reference_vertex_computation
{
    finalize_reference_vertex_computation(const cudaT lambda_range_, const cudaT lambda_offset_,
                                dev_vec_int const& accum_n_branches_per_dim_) :
            lambda_range(lambda_range_), lambda_offset(lambda_offset_),
            accum_n_branches_per_dim_ptr(thrust::raw_pointer_cast(&accum_n_branches_per_dim_[0]))
    {}

    __host__ __device__
    cudaT operator()(const int &reference_vertex, const int &maximum_cube_depth)
    {

        return  reference_vertex * lambda_range / accum_n_branches_per_dim_ptr[maximum_cube_depth + 1] + lambda_offset;
    }

    const cudaT lambda_range;
    const cudaT lambda_offset;
    const int * accum_n_branches_per_dim_ptr;
};


struct finalize_center_vertex_computation
{
    finalize_center_vertex_computation(const cudaT lambda_range_, const cudaT lambda_offset_,
                                dev_vec_int const& accum_n_branches_per_dim_) :
            lambda_range(lambda_range_), lambda_offset(lambda_offset_),
            accum_n_branches_per_dim_ptr(thrust::raw_pointer_cast(&accum_n_branches_per_dim_[0]))
    {}

    __host__ __device__
    cudaT operator()(const int &reference_vertex, const int &maximum_cube_depth)
    {
        // int reference_vertex = thrust::get<0>(t);
        // int maximum_cube_depth  = thrust::get<1>(t) + 1; // = k

        return ((reference_vertex) * lambda_range / accum_n_branches_per_dim_ptr[maximum_cube_depth + 1] +
                (reference_vertex + 1) * lambda_range / accum_n_branches_per_dim_ptr[maximum_cube_depth + 1]) / 2 + lambda_offset;
    }

    const cudaT lambda_range;
    const cudaT lambda_offset;
    const int * accum_n_branches_per_dim_ptr;
};


// Checks if the given number of positive signs is equal to 0 or to upper bound.
// If this is not the case, the given cube contains definitly no fixed point.
// With status, the previous status is taken into account (if it has been recognized already as no fixed point)
struct check_for_no_fixed_point
{
    check_for_no_fixed_point(const int upper_bound_): upper_bound(upper_bound_)
    {}

    __host__ __device__
    bool operator()(const int &val, const bool& status) const
    {
        return ((val == upper_bound) or (val == 0)) or status;
    }

    const int upper_bound;
};


HyperCubes::HyperCubes(const std::vector< std::vector<int>> n_branches_per_depth_,
    const std::vector <std::pair<cudaT, cudaT>> lambda_ranges_) :
    dim(lambda_ranges_.size()),
    n_branches_per_depth(n_branches_per_depth_),
    accum_n_branches_per_dim(HyperCubes::compute_accum_n_branches_per_dim(n_branches_per_depth_, lambda_ranges_.size())),
    accum_n_branches_per_depth(HyperCubes::compute_accum_n_branches_per_depth(n_branches_per_depth_, lambda_ranges_.size())),
    lambda_ranges(lambda_ranges_)
{}

//[ Static functions

thrust::host_vector<thrust::host_vector<int>> HyperCubes::compute_accum_n_branches_per_dim(const std::vector< std::vector<int>> &n_branches_per_depth_, const uint dim_)
{
    thrust::host_vector<thrust::host_vector<int>> accum_n_branches_per_dim_(dim_);
    for(auto dim_index = 0; dim_index < dim_; dim_index++)
    {
        thrust::host_vector<int> accum_n_branches(n_branches_per_depth_.size() + 1);
        accum_n_branches[0] = 1;
        thrust::transform(n_branches_per_depth_.begin(), n_branches_per_depth_.end(), accum_n_branches.begin() + 1, accumulate_n_branches(dim_index, 1));
        accum_n_branches_per_dim_[dim_index] = accum_n_branches;
        // Testing
        // print_range("Accum branches per dim in dim " + std::to_string(dim_index), accum_n_branches.begin(), accum_n_branches.end());
    }
    return accum_n_branches_per_dim_;
}

thrust::host_vector<thrust::host_vector<int>> HyperCubes::compute_accum_n_branches_per_depth(const std::vector< std::vector<int>> &n_branches_per_depth_, const uint dim_)
{
    thrust::host_vector<thrust::host_vector<int>> accum_n_branches_per_depth_(n_branches_per_depth_.size());
    for(auto depth_index = 0; depth_index < n_branches_per_depth_.size(); depth_index++)
    {
        thrust::host_vector<int> accum_n_branches(dim_ + 1);
        accum_n_branches[0] = 1;
        thrust::inclusive_scan(n_branches_per_depth_[depth_index].begin(), n_branches_per_depth_[depth_index].end(), accum_n_branches.begin() + 1, thrust::multiplies<int>());
        accum_n_branches_per_depth_[depth_index] = accum_n_branches;
        // Testing
        // print_range("Accum branches per depth in depth " + std::to_string(depth_index), accum_n_branches.begin(), accum_n_branches.end());
    }
    return accum_n_branches_per_depth_;
}

void HyperCubes::compute_summed_positive_signs_per_cube(dev_vec_bool &velocity_sign_properties_in_dim, dev_vec_int &summed_positive_signs)
{
    // Initialize a vectors for sign checks
    auto total_number_of_cubes = summed_positive_signs.size();
    auto total_number_of_vertices = velocity_sign_properties_in_dim.size();
    if(total_number_of_cubes != 0)
    {
        auto number_of_vertices_per_cube = int(total_number_of_vertices / total_number_of_cubes); // = pow(2, dim)

        dev_vec_int indices_of_summed_positive_signs(total_number_of_vertices);

        // Necessary that reduce by key works (cannot handle mixture of bool and integer), ToDo: Alternative solution??
        dev_vec_int int_velocity_sign_properties_in_dim(velocity_sign_properties_in_dim.begin(), velocity_sign_properties_in_dim.end());

        /*Use iterators to transform the linear index into a row index -> the final iterator repeats the
        * row indices (0 to pow(2, dim)-1) total_number_of_cubes times, i.e.: 0 0 0 0 0 0 0 0 1 1 1 1 1 1 1 1
        * These are then used as a mask to define which signs in vertex_velocity should be summed up.
        * indices_of_summed_positive_signs contains the keys for the mask, i.e. (0, 1, 2, etc.) and
        * summed_positive_signs the corresponding sum per key. */
        // Sum positive signs
        thrust::reduce_by_key
                (thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                                 linear_index_to_row_index<int>(number_of_vertices_per_cube)),
                 thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                                 linear_index_to_row_index<int>(number_of_vertices_per_cube)) +
                 (number_of_vertices_per_cube * total_number_of_cubes),
                 int_velocity_sign_properties_in_dim.begin(),
                 indices_of_summed_positive_signs.begin(),
                 summed_positive_signs.begin(),
                 thrust::equal_to<int>(),
                 thrust::plus<int>());
    }
}

//]

GridComputationWrapper HyperCubes::project_coordinates_on_expanded_cube_and_depth_per_cube_indices(odesolver::DevDatC coordinates, bool coordinates_on_grid, int depth) const // no reference since coordinates is changed within this function
{
    if(depth == -1)
        depth = n_branches_per_depth.size() - 1;
    else if(depth > n_branches_per_depth.size() - 1)
    {
        std::cout << "\nERROR: Maximum number of branches per depth " << n_branches_per_depth.size() << " do not coincide with depth " << depth <<  std::endl;
        std::exit(EXIT_FAILURE);
    }

    int tnoc = coordinates.n_elems(); // total number of cubes
    GridComputationWrapper grcompwrap(tnoc, depth + 1, depth);

    for(auto dim_index = 0; dim_index < dim; dim_index++)
    {
        cudaT lambda_dim_range = (lambda_ranges[dim_index].second - lambda_ranges[dim_index].first);
        cudaT lambda_range_left = lambda_ranges[dim_index].first;

        cudaT lambda_offset;
        if(coordinates_on_grid) {
            lambda_offset = 0.5*lambda_dim_range/accum_n_branches_per_dim[dim_index][depth + 1]; // For avoidance of rounding errors -> corresponds to half of the width of the smallest considered cube
        }
        else  {
            lambda_offset = 0.0;
        }

        dev_vec temp_coordinates(coordinates[dim_index].begin(), coordinates[dim_index].end());

        // Shift coordinates to reference system (most left coordinate == 0)
        thrust::transform(temp_coordinates.begin(), temp_coordinates.end(), temp_coordinates.begin(),
        [lambda_range_left] __host__ __device__(const cudaT &coor) { return coor - lambda_range_left; });

        for(auto depth_index = 0; depth_index < grcompwrap.expanded_cube_indices_.dim_size(); depth_index++)
        {

            cudaT delta_lambda = lambda_dim_range / accum_n_branches_per_dim[dim_index][depth_index + 1]; // corresponds to the width of the considered cube

            // Compute axis indices
            dev_vec_int axis_index(tnoc, 0); // corresponds to the index of the considered axis
            thrust::transform(temp_coordinates.begin(), temp_coordinates.end(), axis_index.begin(),
                              compute_axis_index_of_coor(lambda_offset, delta_lambda, n_branches_per_depth[depth_index][dim_index]));

            // Shift coordinates to corresponding new reference system in considered depth
            thrust::transform(temp_coordinates.begin(), temp_coordinates.end(), axis_index.begin(),
                              temp_coordinates.begin(),
            [delta_lambda] __host__ __device__ (const cudaT &coor, const cudaT &cube_index)
            {
                return coor - (cube_index * delta_lambda);
            });

            // Add axis index to expanded cube indices
            thrust::transform(axis_index.begin(), axis_index.end(),
                              grcompwrap.expanded_cube_indices_[depth_index].begin(),
                              grcompwrap.expanded_cube_indices_[depth_index].begin(),
                              sum_axis_indices_to_cube_index(accum_n_branches_per_depth[depth_index][dim_index]));
        }
    }

    return grcompwrap;
}

GridComputationWrapper HyperCubes::project_leaves_on_expanded_cube_and_depth_per_cube_indices(std::vector<Leaf*> &leaves, int depth) const
{
    if(depth == -1)
        depth = n_branches_per_depth.size() - 1;
    else if(depth > n_branches_per_depth.size() - 1)
    {
        std::cout << "\nERROR: Maximum number of branches per depth " << n_branches_per_depth.size() << " do not coincide with depth " << depth <<  std::endl;
        std::exit(EXIT_FAILURE);
    }

    int tnoc = leaves.size();

    GridComputationWrapper grcompwrap(tnoc, depth +1 , depth);

    thrust::host_vector<int> host_expanded_cube_indices ((depth + 1) * tnoc, 0);
    for(auto depth_index = 0; depth_index < grcompwrap.expanded_cube_indices_.dim_size(); depth_index++)
    {
        thrust::transform(thrust::host, leaves.begin(), leaves.end(), host_expanded_cube_indices.begin() + tnoc * depth_index, [depth_index] (const Leaf * leaf) { return leaf->get_ith_cube_depth_index(depth_index); });
    }
    grcompwrap.expanded_cube_indices_.fill_by_vec(host_expanded_cube_indices);
    return grcompwrap;
}

void HyperCubes::compute_reference_vertices(odesolver::DevDatC &reference_vertices, GridComputationWrapper &grcompwrap)
{
    for(auto dim_index = 0; dim_index < dim; dim_index++) {
        compute_reference_vertex_in_dim(reference_vertices[dim_index], grcompwrap, dim_index);

        // Compute delta range
        cudaT lambda_dim_range = (lambda_ranges[dim_index].second - lambda_ranges[dim_index].first);
        cudaT lambda_offset = lambda_ranges[dim_index].first;

        // Finalize computation of the device reference vertex
        thrust::transform(reference_vertices[dim_index].begin(), reference_vertices[dim_index].end(), grcompwrap.expanded_depth_per_cube_.begin(), reference_vertices[dim_index].begin(),
                          finalize_reference_vertex_computation(lambda_dim_range, lambda_offset, accum_n_branches_per_dim[dim_index]));
    }
    vertex_mode = ReferenceVertices;;
}

odesolver::DevDatC HyperCubes::compute_reference_vertices(GridComputationWrapper &grcompwrap)
{
    // Initialize reference_vertices
    auto total_number_of_cubes = grcompwrap.expanded_depth_per_cube_.size();
    auto reference_vertices = odesolver::DevDatC (dim, total_number_of_cubes);
    
    // Compute reference_vertices
    compute_reference_vertices(reference_vertices, grcompwrap);

    return std::move(reference_vertices);
}

void HyperCubes::compute_vertices(odesolver::DevDatC &vertices, GridComputationWrapper &grcompwrap, int total_number_of_cubes)
{
    if(total_number_of_cubes == 0)
        total_number_of_cubes = grcompwrap.expanded_depth_per_cube_.size();

    for(auto dim_index = 0; dim_index < dim; dim_index++)
    {
        // Generate device vector of reference vertices for each vector
        odesolver::DevDatC reference_vertices_wrapper(1, total_number_of_cubes, 0.0);
        odesolver::DimensionIteratorC& reference_vertices_ = reference_vertices_wrapper[0];
        compute_reference_vertex_in_dim(reference_vertices_, grcompwrap, dim_index);

        // Testing -> Can be used as test without regarding the correct reference vertices
        // print_range("Reference vertices in dimension " + std::to_string(dim_index + 1), reference_vertices_.begin(), reference_vertices_.end());
        /* // Compute delta ranges
         * cudaT lambda_dim_range = (lambda_ranges[dim_index].second - lambda_ranges[dim_index].first);
         * cudaT lambda_offset = lambda_ranges[dim_index].first;
         * thrust::transform(reference_vertices_.begin(), reference_vertices_.end(), grcompwrap.expanded_depth_per_cube_.begin(), reference_vertices_.begin(),
                          finalize_reference_vertex_computation(lambda_dim_range, lambda_offset, accum_n_branches_per_dim[dim_index])) */

        // Preparations for the expansion to vertices

        // Repeat reference vertex according to the number of vertices per cube
        repeated_range<dev_iterator> rep_ref_vertex_iterator(
            reference_vertices_.begin(),
            reference_vertices_.end(),
            pow(2, dim));

        // Repeat maximum depth values according to the number of vertices per cube
        repeated_range<dev_iterator_int> rep_ref_depth_per_cube_iterator(
            grcompwrap.expanded_depth_per_cube_.begin(),
            grcompwrap.expanded_depth_per_cube_.begin() + total_number_of_cubes,
            pow(2, dim)
        );

        // Compute inner cube offset
        dev_vec_bool inner_vertex_coors(pow(2, dim)); // Computation can be shifted to global function (-> apropriate??)
        thrust::tabulate(inner_vertex_coors.begin(), inner_vertex_coors.end(), compute_inner_vertex_coor(dim_index));
        tiled_range<dev_iterator_bool> rep_inner_vertex_coors(inner_vertex_coors.begin(), inner_vertex_coors.end(), total_number_of_cubes);

        // Testing
        // print_range("Inner coors", inner_vertex_coors.begin(), inner_vertex_coors.end());

        // Finalize

        // Compute delta range
        cudaT lambda_dim_range = (lambda_ranges[dim_index].second - lambda_ranges[dim_index].first);
        cudaT lambda_offset = lambda_ranges[dim_index].first;

        // Finalize computation of device vertex
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(rep_ref_vertex_iterator.begin(), rep_ref_depth_per_cube_iterator.begin(), rep_inner_vertex_coors.begin(), vertices[dim_index].begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(rep_ref_vertex_iterator.end(), rep_ref_depth_per_cube_iterator.end(), rep_inner_vertex_coors.end(), vertices[dim_index].begin() + total_number_of_cubes * pow(2, dim))),
                         finalize_vertex_computation(lambda_dim_range, lambda_offset, accum_n_branches_per_dim[dim_index]));

        // Testing
         /* if(monitor)
            print_range("Vertices in dimension " + std::to_string(dim_index + 1), vertices[dim_index].begin(), vertices[dim_index].end()); */
    }

    vertex_mode = CubeVertices;
}

odesolver::DevDatC HyperCubes::compute_vertices(GridComputationWrapper &grcompwrap)
{
    // Initialize vertices
    auto total_number_of_cubes = grcompwrap.expanded_depth_per_cube_.size();
    auto vertices = odesolver::DevDatC(dim, total_number_of_cubes * pow(2, dim));
    
    // Compute vertices
    compute_vertices(vertices, grcompwrap);

    return std::move(vertices);
}

void HyperCubes::compute_cube_center_vertices(odesolver::DevDatC &center_vertices, GridComputationWrapper &grcompwrap)
{
    for (auto dim_index = 0; dim_index < dim; dim_index++) {
        // Generate device vector of reference vertices for each vector
        compute_reference_vertex_in_dim(center_vertices[dim_index], grcompwrap, dim_index);

        // Finalize

        // Compute delta range
        cudaT lambda_dim_range = (lambda_ranges[dim_index].second - lambda_ranges[dim_index].first);
        cudaT lambda_range_left = lambda_ranges[dim_index].first;

        // Finalize computation of device center vertex
        thrust::transform(center_vertices[dim_index].begin(), center_vertices[dim_index].end(),
                          grcompwrap.expanded_depth_per_cube_.begin(), center_vertices[dim_index].begin(),
                          finalize_center_vertex_computation(lambda_dim_range, lambda_range_left,
                                                             accum_n_branches_per_dim[dim_index]));

        // Testing
        if (monitor)
            print_range("Cube center vertices in dimension " + std::to_string(dim_index + 1),
                        center_vertices[dim_index].begin(), center_vertices[dim_index].end());
    }

    vertex_mode = CenterVertices;
}

odesolver::DevDatC HyperCubes::compute_cube_center_vertices(GridComputationWrapper &grcompwrap)
{
    // Initialize center_vertices
    auto total_number_of_cubes = grcompwrap.expanded_depth_per_cube_.size();
    auto center_vertices = odesolver::DevDatC(dim, total_number_of_cubes);
        
    // Compute center_vertices
    compute_cube_center_vertices(center_vertices, grcompwrap);

    return std::move(center_vertices);
}

/* void HyperCubes::determine_vertex_velocities(FlowEquationsWrapper * flow_equations)
{
    vertex_velocities = compute_vertex_velocities(vertices, flow_equations); */
    // Testing
    /* if(monitor)
        for(auto dim_index = 0; dim_index < dim; dim_index++)
            print_range("Vertex velocities in dimension " + std::to_string(dim_index + 1), vertex_velocities[dim_index].begin(), vertex_velocities[dim_index].end()); */
// }

thrust::host_vector<int> HyperCubes::determine_potential_fixed_points(odesolver::DevDatC& vertex_velocities, int total_number_of_cubes)
{
    if (vertex_mode != CubeVertices and vertex_mode != CenterVertices)
    {
        std::cout << "\nERROR: Wrong vertex mode for computation of potential fix points" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    if(total_number_of_cubes == 0)
        total_number_of_cubes = int(vertex_velocities.n_elems() / pow(2, dim));

    auto number_of_vertices_ = vertex_velocities.n_elems(); // to avoid a pass of this within the lambda capture
    thrust::host_vector<dev_vec_bool> velocity_sign_properties(dim);
    thrust::generate(velocity_sign_properties.begin(), velocity_sign_properties.end(), [number_of_vertices_]() { return dev_vec_bool (number_of_vertices_, false); });

    // Initial potential fixed points -> at the beginning all cubes contain potential fixed points ( false = potential fixed point )
    dev_vec_bool pot_fixed_points(total_number_of_cubes, false);
    for(auto dim_index = 0; dim_index < dim; dim_index ++)
    {
        // Turn vertex_velocities into an array with 1.0 and 0.0 for change in sign
        thrust::transform(vertex_velocities[dim_index].begin(), vertex_velocities[dim_index].end(), velocity_sign_properties[dim_index].begin(), greater_than_zero());

        // Initialize a vector for sign checks
        dev_vec_int summed_positive_signs(total_number_of_cubes, 0); // Contains the sum of positive signs within each cube
        HyperCubes::compute_summed_positive_signs_per_cube(velocity_sign_properties[dim_index], summed_positive_signs);

        // Testing
        if(monitor)
            print_range("Summed positive signs in dim " + std::to_string(dim_index), summed_positive_signs.begin(), summed_positive_signs.end());

        // Check if the sign has changed in this component (dimension), takes the previous status into account
        thrust::transform(summed_positive_signs.begin(), summed_positive_signs.end(), pot_fixed_points.begin(), pot_fixed_points.begin(), check_for_no_fixed_point(pow(2, dim)));
    }

    // Genereate mock fixed points
    //srand(13);
    //thrust::generate(thrust::host, pot_fixed_points.begin(), pot_fixed_points.end(), []() { return 0; } ); // rand() % 8

    // Test output
    /* std::cout << "Potential fixed points in linearized vertex velocities: " << std::endl;
    int i = 0;
    for(const auto &elem : pot_fixed_points) {
        std::cout << i << ": " << elem << " - ";
        i++;
    }
    std::cout << std::endl; */

    // Reduce on indices with potential fixed points (filter the value with pot_fixed_points==True) // (offset iterator + 1)  -> not used anymore (why initially used??)
    dev_vec_int indices_of_pot_fixed_points(total_number_of_cubes);
    auto last_potential_fixed_point_iterator = thrust::remove_copy_if(
            thrust::make_counting_iterator(0),
            thrust::make_counting_iterator(total_number_of_cubes),
            pot_fixed_points.begin(), // Works as mask for values that should be copied (checked if identity is fulfilled)
            indices_of_pot_fixed_points.begin(),
            thrust::identity<int>());

    // Store valid indices of potential fixed points in host_indices_of_pot_fixed_points
    thrust::host_vector<int> host_indices_of_pot_fixed_points(indices_of_pot_fixed_points.begin(), last_potential_fixed_point_iterator);
    // indices_of_pot_fixed_points.resize(last_potential_fixed_point_iterator - indices_of_pot_fixed_points.begin());  -> alternative way to do this
    // host_indices_of_pot_fixed_points = indices_of_pot_fixed_points;

    // Test output
    /* std::cout << "Indices of potential fixed points: " << std::endl;
    for(auto &elem : host_indices_of_pot_fixed_points)
        std::cout << elem << " ";
    std::cout << std::endl; */

    return host_indices_of_pot_fixed_points;
}

// Getter functions

/* const odesolver::DevDatC& HyperCubes::get_vertex_velocities() const
{
    return vertex_velocities;
} */

const std::vector<std::vector<int>>& HyperCubes::get_n_branches_per_depth() const
{
    return n_branches_per_depth;
}

const std::vector<std::pair<cudaT, cudaT>>& HyperCubes::get_lambda_ranges() const
{
    return lambda_ranges;
}

// Test function

// Todo!!
// Function for testing if project_coordinates_on_expanded_cube_and_depth_per_cube_indices works
void HyperCubes::test_projection()
{
    /* // Testing projection of coordinates on expanded cube and depth per cube indices
    dev_ptrvec_vec_int * expanded_cube_indices_ptr = nullptr;
    dev_vec_int * expanded_depth_per_cube_ptr = nullptr;
    std::tie(expanded_cube_indices_ptr, expanded_depth_per_cube_ptr) = project_coordinates_on_expanded_cube_and_depth_per_cube_indices(vertices, true, 0);
    if(monitor)
    {
        auto i = 0;
        for(auto elem : *expanded_cube_indices_ptr)
        {
            print_range("Recheck Expanded cube indices after filling with individual cube indices in depth " + std::to_string(i), elem->begin(), elem->end());
            i++;
        }
        print_range("Recheck Expanded depth per node", expanded_depth_per_cube_ptr->begin(), expanded_depth_per_cube_ptr->end());
    }

    // Reduce on cube reference indices
    const uint8_t dim_ = dim;
    auto i = 0;
    for(auto depth_index = 0; depth_index < expanded_cube_indices_ptr->size(); depth_index++)
    {
        auto last_expanded_cube_index_iterator = thrust::remove_copy_if(
                (*expanded_cube_indices_ptr)[depth_index]->begin(),
                (*expanded_cube_indices_ptr)[depth_index]->end(),
                thrust::make_counting_iterator(0), // Works as mask for values that should be copied (checked if identity is fulfilled)
                (*expanded_cube_indices_ptr)[depth_index]->begin(),
        [dim_] __host__ __device__ (const int &val) { return val % int(pow(2,dim_)); });

        (*expanded_cube_indices_ptr)[depth_index]->resize(last_expanded_cube_index_iterator - (*expanded_cube_indices_ptr)[depth_index]->begin());
        if(monitor)
            print_range("Reduced set in depth " + std::to_string(i), (*expanded_cube_indices_ptr)[depth_index]->begin(), (*expanded_cube_indices_ptr)[depth_index]->end());
        i++;
    }

    auto last_expanded_depth_index_iterator = thrust::remove_copy_if(
            expanded_depth_per_cube_ptr->begin(),
            expanded_depth_per_cube_ptr->end(),
            thrust::make_counting_iterator(0), // Works as mask for values that should be copied (checked if identity is fulfilled)
            expanded_depth_per_cube_ptr->begin(),
    [dim_] __host__ __device__ (const int &val) { return val % int(pow(2,dim_)); });

    expanded_depth_per_cube_ptr->resize(last_expanded_depth_index_iterator - expanded_depth_per_cube_ptr->begin());

    if(monitor)
        print_range("Reduced expanded depth", expanded_depth_per_cube_ptr->begin(), expanded_depth_per_cube_ptr->end());

    compute_vertices(*expanded_cube_indices_ptr, *expanded_depth_per_cube_ptr);

    // Free memory
    delete expanded_depth_per_cube_ptr;
    //delete (*expanded_cube_indices_ptr)[0];
    thrust::for_each(expanded_cube_indices_ptr->begin(), expanded_cube_indices_ptr->end(), [] (dev_vec_int *elem) { delete elem; }); */
}

// Include this function into hypercubes?? # Currently not used!!
/* thrust::host_vector< dev_vec_bool* > * get_velocity_sign_properties(std::vector< dev_vec* > &vertex_velocities)
{
    const uint dim = vertex_velocities.size();
    auto number_of_coordinates_ = vertex_velocities[0]->size(); // to avoid a pass of this within the lambda capture
    auto * velocity_sign_properties_ptr = new thrust::host_vector< dev_vec_bool* > (dim);
    thrust::generate(velocity_sign_properties_ptr->begin(), velocity_sign_properties_ptr->end(), [number_of_coordinates_]() { return new dev_vec_bool (number_of_coordinates_, false); });
    for(auto dim_index = 0; dim_index < dim; dim_index ++)
    {
        // Turn vertex_velocities into an array with 1.0 and 0.0 for change in sign
        thrust::transform(vertex_velocities[dim_index]->begin(), vertex_velocities[dim_index]->end(), (*velocity_sign_properties_ptr)[dim_index]->begin(), greater_than_zero());
    }
    return velocity_sign_properties_ptr;
} */


// Private functions

void HyperCubes::compute_reference_vertex_in_dim(odesolver::DimensionIteratorC &reference_vertices_, GridComputationWrapper &grcompwrap, int dim_index, int total_number_of_cubes, int maximum_depth) const
{
    if(total_number_of_cubes == 0)
        total_number_of_cubes = grcompwrap.expanded_depth_per_cube_.size();
    if(maximum_depth == 0)
        maximum_depth = grcompwrap.expanded_cube_indices_.dim_size();

    for(auto depth_index = 0; depth_index < maximum_depth; depth_index++)
    {
        int accum_n_branch_per_depth = accum_n_branches_per_depth[depth_index][dim_index];
        int n_branch_per_depth = n_branches_per_depth[depth_index][dim_index];
        int depth_weight_divisor = accum_n_branches_per_dim[dim_index][depth_index + 1];
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(grcompwrap.expanded_cube_indices_[depth_index].begin(), grcompwrap.expanded_depth_per_cube_.begin(), reference_vertices_.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(grcompwrap.expanded_cube_indices_[depth_index].begin() + total_number_of_cubes, grcompwrap.expanded_depth_per_cube_.begin() + total_number_of_cubes, reference_vertices_.end())),
                         compute_depth_vertex_coor_weight(n_branch_per_depth, accum_n_branch_per_depth, depth_weight_divisor, accum_n_branches_per_dim[dim_index]));
    }
}