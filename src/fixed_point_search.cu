#include "hip/hip_runtime.h"
#include "../include/odesolver/fixed_point_search.hpp"

// FixedPointSearch Constructors

FixedPointSearch::FixedPointSearch(const json params, const PathParameters path_parameters) : FRGVisualizationParameters(params, path_parameters),
                                                    dim_(get_entry<int>("dim")),
                                                    maximum_recursion_depth_(get_entry<int>("maximum_recursion_depth")),
                                                    k_(get_entry<cudaT>("k"))
{
    auto n_branches_per_depth = get_entry<json>("n_branches_per_depth");
    auto lambda_ranges = get_entry<json>("lambda_ranges");

    std::transform(n_branches_per_depth.begin(), n_branches_per_depth.end(), std::back_inserter(n_branches_per_depth_),
                   [] (json &dat) { return dat.get< std::vector<int> >(); });
    std::transform(lambda_ranges.begin(), lambda_ranges.end(), std::back_inserter(lambda_ranges_),
                   [] (json &dat) { return dat.get< std::pair<cudaT, cudaT> >(); });
    
    flow_equations_ = FlowEquationsWrapper::make_flow_equation(path_parameters_.theory_);

    Node * root_node_ptr = new Node(0, compute_internal_end_index(n_branches_per_depth_[0]), std::vector< int >{});
    buffer_ = Buffer(root_node_ptr);

    // Tests
    if (n_branches_per_depth_.size() != maximum_recursion_depth_)
    {
        std::cout << "\nERROR: Maximum recursion depth " << maximum_recursion_depth_
                  << " and number of branches per depth " << n_branches_per_depth_.size()
                  << " do not coincide" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    for (const auto &n_branches : n_branches_per_depth_)
    {
        if (n_branches.size() != dim_) {
            std::cout << "\nERROR: Number of branches per depth " << n_branches.size() << " do not coincide with dimension " << dim_ <<  std::endl;
            std::exit(EXIT_FAILURE);
        }
    }

    if(lambda_ranges_.size() != dim_)
    {
        std::cout << "\nERROR: Number of lambda ranges " << lambda_ranges_.size() << " do not coincide with dimension" << dim_ << std::endl;
        std::exit(EXIT_FAILURE);
    }

    if(flow_equations_->get_dim() != dim_)
    {
        std::cout << "\nERROR: Dimensions and number of flow equation do not coincide" << dim_ << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

FixedPointSearch::FixedPointSearch(const std::string theory,
                                                       const std::string mode_type,
                                                       const std::string results_dir,
                                                       const std::string root_dir,
                                                       const bool relative_path) : FixedPointSearch(
                                                               param_helper::fs::read_parameter_file(
        root_dir + "/" + theory + "/" + results_dir + "/", "config", relative_path),
        PathParameters(theory, mode_type, root_dir, relative_path))
{}

FixedPointSearch::FixedPointSearch(
        const std::string theory,
        const int maximum_recursion_depth,
        const std::vector< std::vector<int> > n_branches_per_depth,
        const std::vector <std::pair<cudaT, cudaT> > lambda_ranges,
        const std::string mode,
        const std::string root_dir,
        const bool relative_path
) : FixedPointSearch(
        json {{"maximum_recursion_depth", maximum_recursion_depth},
              {"n_branches_per_depth", n_branches_per_depth},
              {"lambda_ranges", lambda_ranges},
              {"mode", mode}},
        PathParameters(theory, mode, root_dir, relative_path)
)
{}


FixedPointSearch::ClusterParameters::ClusterParameters(const json params) : Parameters(params),
                                        maximum_expected_number_of_clusters_(get_entry<uint>("maximum_expected_number_of_clusters")),
                                        upper_bound_for_min_distance_(get_entry<double>("upper_bound_for_min_distance")),
                                        maximum_number_of_iterations_(get_entry<uint>("maximum_number_of_iterations"))
{}

FixedPointSearch::ClusterParameters::ClusterParameters(
        const uint maximum_expected_number_of_clusters,
        const double upper_bound_for_min_distance,
        const uint maximum_number_of_iterations
) : ClusterParameters(
        json {{"maximum_expected_number_of_clusters", maximum_expected_number_of_clusters},
              {"upper_bound_for_min_distance", upper_bound_for_min_distance},
              {"maximum_number_of_iterations", maximum_number_of_iterations}}
)
{}

// Main functions

void FixedPointSearch::find_fixed_point_solutions()
{
    auto c = 0;
    while(c < computation_parameters_.maximum_number_of_gpu_calls_ and buffer_.len() > 0)
    {
        std::cout << "\n\n######### New computation round: " << c <<  "#########" << std::endl;
        NodeCounter<Node>::number_of_alive_nodes_per_depth();
        std::cout << std::endl;
        run_gpu_computing_task();
        c++;
    }
}

void FixedPointSearch::cluster_solutions_to_fixed_points(const uint maximum_expected_number_of_clusters,
        const double upper_bound_for_min_distance,
        const uint maximum_number_of_iterations)
{
    // Compute vertices of solutions
    HyperCubes solution_cubes(k_, n_branches_per_depth_, lambda_ranges_);

    GridComputationWrapper grcompwrap = solution_cubes.project_leaves_on_expanded_cube_and_depth_per_cube_indices(solutions_);
    solution_cubes.compute_cube_center_vertices(grcompwrap);

    // Get center vertices
    const odesolver::DevDatC potential_fixed_points = solution_cubes.get_vertices();

    // Cluster center vertices
    fixed_points_ = cluster_device_data(
            maximum_expected_number_of_clusters,
            upper_bound_for_min_distance,
            potential_fixed_points,
            maximum_number_of_iterations
            );
}

void FixedPointSearch::cluster_solutions_to_fixed_points_from_parameters(const FixedPointSearch::ClusterParameters cluster_parameters)
{
    cluster_solutions_to_fixed_points(
            cluster_parameters.maximum_expected_number_of_clusters_,
            cluster_parameters.upper_bound_for_min_distance_,
            cluster_parameters.maximum_number_of_iterations_);
}

void FixedPointSearch::cluster_solutions_to_fixed_points_from_file()
{
    auto cluster_params = get_entry<json>("cluster");
    auto cluster_parameters = FixedPointSearch::ClusterParameters(cluster_params);
    cluster_solutions_to_fixed_points_from_parameters(cluster_parameters);
}

std::vector<Leaf*> FixedPointSearch::get_solutions()
{
    return solutions_;
}

odesolver::DevDatC FixedPointSearch::get_fixed_points() const
{
    return fixed_points_;
}


/* ## ToDo: Reinclude - Commented during reodering 
void FixedPointSearch::compute_and_write_fixed_point_characteristics_to_file(std::string dir)
{
    // ToDo: If fixed_points are not defined - Try to load them file first, otherwise throw error

    CoordinateOperatorParameters coordinate_operator_parameters = CoordinateOperatorParameters::from_parameters(
            path_parameters.theory,
            {},
            "evaluate",
            path_parameters.root_dir,
            path_parameters.relative_path);
    CoordinateOperator fixed_point_evaluator(coordinate_operator_parameters);
    fixed_point_evaluator.set_raw_coordinates(fixed_points);

    fixed_point_evaluator.compute_velocities();
    fixed_point_evaluator.compute_jacobians_and_eigendata();

    fixed_point_evaluator.write_characteristics_to_file(dir);
} */

void FixedPointSearch::write_solutions_to_file(std::string dir) const
{
    json j;
    for(auto &sol: solutions_)
        j.push_back(sol->to_json());
    param_helper::fs::write_parameter_file(json {{"number_of_solutions", solutions_.size()}, {"solutions", j}}, path_parameters_.get_base_path() + dir + "/", "solutions", path_parameters_.relative_path_);
}

void FixedPointSearch::load_solutions_from_file(std::string dir)
{
    clear_solutions();

    json j = param_helper::fs::read_parameter_file(path_parameters_.get_base_path() + dir + "/", "solutions", path_parameters_.relative_path_);
    solutions_.reserve(j["number_of_solutions"].get< int >());
    for(auto &sol: j["solutions"])
        solutions_.push_back(new Leaf(sol["cube_indices"].get< std::vector<int> >()));
    std::cout << "solutions loaded" << std::endl;
}

void FixedPointSearch::write_fixed_points_to_file(std::string dir) const
{
    json j;
    auto transposed_fixed_points = fixed_points_.transpose_device_data();
    for(auto &fixed_point : transposed_fixed_points)
        j.push_back(fixed_point);
    param_helper::fs::write_parameter_file(json {{"number_of_fixed_points", transposed_fixed_points.size()}, {"fixed_points", j}}, path_parameters_.get_base_path() + "/" + dir + "/", "fixed_points", path_parameters_.relative_path_);
}

void FixedPointSearch::load_fixed_points_from_file(std::string dir)
{
    json j = param_helper::fs::read_parameter_file(path_parameters_.get_base_path() + dir + "/", "fixed_points", path_parameters_.relative_path_);
    std::vector < std::vector<double> > fixed_points;
    fixed_points.reserve(j["number_of_fixed_points"].get< int >());
    for(auto &sol: j["fixed_points"])
        fixed_points.push_back(sol.get< std::vector<double> >());
    fixed_points_ = odesolver::DevDatC(fixed_points);
    std::cout << "fixed points loaded" << std::endl;
}

// Iterate over nodes and generate new nodes based on the indices of pot fixed points
std::tuple< std::vector<Node* >, std::vector< Leaf* > > FixedPointSearch::generate_new_nodes_and_leaves(const thrust::host_vector<int> &host_indices_of_pot_fixed_points, const std::vector< Node* > &nodes)
{
    std::vector< Node* > new_nodes;
    std::vector< Leaf* > new_leaves;

    // No potential fixed points have been found
    if(host_indices_of_pot_fixed_points.size() > 0)
    {
        // Initial conditions
        auto pot_fixed_point_iterator = host_indices_of_pot_fixed_points.begin();
        int index_offset = 0;

        // Iterate over nodes
        for(const auto &node : nodes)
        {
            // Get first potential fix point -> is defined with respect to 0
            int index_of_pot_fixed_point = *pot_fixed_point_iterator - index_offset; // (-1 to undo offset) -> not used anymore (why initially used??)

            // Fix points have been found in node
            if(index_of_pot_fixed_point < node->get_n_cubes())
            {
                // Inspect fixed points
                while(index_of_pot_fixed_point < node->get_n_cubes() and pot_fixed_point_iterator != host_indices_of_pot_fixed_points.end())
                {
                    // Compute parent node indices
                    std::vector<int> parent_cube_indices(node->get_parent_cube_indices());
                    parent_cube_indices.push_back(index_of_pot_fixed_point + node->get_internal_start_index());

                    // Generate new nodes
                    if(node->get_depth() + 1 < maximum_recursion_depth_) {
                        new_nodes.push_back(
                                new Node(0, compute_internal_end_index(n_branches_per_depth_[node->get_depth() + 1]),
                                         parent_cube_indices));
                    }
                    else // Found solution -> Generate new leaf
                    {
                        new_leaves.push_back(new Leaf(parent_cube_indices));
                    }
                    // Update
                    pot_fixed_point_iterator++;
                    index_of_pot_fixed_point = *pot_fixed_point_iterator - index_offset; // (-1 to undo offset) -> not used anymore (why initially used??)
                }
            }

            // Update index offset
            index_offset += node->get_n_cubes();
        }

        assert(host_indices_of_pot_fixed_points.size() ==  (new_nodes.size() + new_leaves.size()) && "Number of new nodes and number of potential fixed points do not coincide");
    }
    return std::make_tuple(new_nodes, new_leaves);
}

void FixedPointSearch::run_gpu_computing_task()
{
    std::vector< Node* > nodes_to_be_computed;
    int total_number_of_cubes = 0;
    int maximum_depth = 0;

    // Get nodes for the gpu from buffer
    std::tie(nodes_to_be_computed, total_number_of_cubes, maximum_depth) = buffer_.get_first_nodes(computation_parameters_.number_of_cubes_per_gpu_call_);

    if(monitor) {
        std::cout << "\n### Nodes for the qpu: " << nodes_to_be_computed.size() << ", total number of cubes: "
                  << total_number_of_cubes << std::endl;
        buffer_.get_nodes_info(nodes_to_be_computed);
    }

    HyperCubes hypercubes(k_, n_branches_per_depth_, lambda_ranges_);

    // Use helper class to perform gpu tasks on nodes
    GridComputationWrapper grcompwrap = hypercubes.generate_and_linearize_nodes(total_number_of_cubes, maximum_depth, nodes_to_be_computed);

    // Compute the actual vertices by first expanding each cube according to the number of vertices to
    // a vector of reference vertices of length total_number_of_cubes*dim and then computing the indices
    hypercubes.compute_vertices(grcompwrap);

    // hypercubes.test_projection();

    // Compute vertex velocities
    hypercubes.determine_vertex_velocities(flow_equations_);

    // Determine potential fix points
    thrust::host_vector<int> host_indices_of_pot_fixed_points = hypercubes.determine_potential_fixed_points();

    // Generate new nodes and derive solutions based on nodes and indices of potential fixed points
    std::vector< Node* > new_nodes;
    std::vector< Leaf* > new_leaves;
    std::tie(new_nodes, new_leaves) = generate_new_nodes_and_leaves(host_indices_of_pot_fixed_points, nodes_to_be_computed);
    buffer_.add_nodes(new_nodes);
    solutions_.insert(solutions_.end(), new_leaves.begin(), new_leaves.end());

    if(monitor) {
        std::cout << "\n### New nodes" << std::endl;
        buffer_.get_nodes_info(new_nodes);
    }

    // Delete evaluated nodes
    for(auto &node : nodes_to_be_computed)
    {
        --NodeCounter<Node>::objects_alive[node->get_depth()];
        delete node;
    }
}

void FixedPointSearch::clear_solutions()
{
    for(auto &sol: solutions_)
        delete sol;
    solutions_.clear();
}
