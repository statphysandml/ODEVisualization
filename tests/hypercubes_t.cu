#include "hip/hip_runtime.h"
// Todo!!
// Function for testing if project_coordinates_on_expanded_cube_and_depth_per_cube_indices works
void HyperCubes::test_projection()
{
    /* // Testing projection of coordinates on expanded cube and depth per cube indices
    dev_ptrvec_vec_int * expanded_cube_indices_ptr = nullptr;
    dev_vec_int * expanded_depth_per_cube_ptr = nullptr;
    std::tie(expanded_cube_indices_ptr, expanded_depth_per_cube_ptr) = project_coordinates_on_expanded_cube_and_depth_per_cube_indices(vertices, true, 0);
    if(monitor)
    {
        auto i = 0;
        for(auto elem : *expanded_cube_indices_ptr)
        {
            print_range("Recheck Expanded cube indices after filling with individual cube indices in depth " + std::to_string(i), elem->begin(), elem->end());
            i++;
        }
        print_range("Recheck Expanded depth per node", expanded_depth_per_cube_ptr->begin(), expanded_depth_per_cube_ptr->end());
    }

    // Reduce on cube reference indices
    const uint8_t dim_ = dim;
    auto i = 0;
    for(auto depth_index = 0; depth_index < expanded_cube_indices_ptr->size(); depth_index++)
    {
        auto last_expanded_cube_index_iterator = thrust::remove_copy_if(
                (*expanded_cube_indices_ptr)[depth_index]->begin(),
                (*expanded_cube_indices_ptr)[depth_index]->end(),
                thrust::make_counting_iterator(0), // Works as mask for values that should be copied (checked if identity is fulfilled)
                (*expanded_cube_indices_ptr)[depth_index]->begin(),
        [dim_] __host__ __device__ (const int &val) { return val % int(pow(2,dim_)); });

        (*expanded_cube_indices_ptr)[depth_index]->resize(last_expanded_cube_index_iterator - (*expanded_cube_indices_ptr)[depth_index]->begin());
        if(monitor)
            print_range("Reduced set in depth " + std::to_string(i), (*expanded_cube_indices_ptr)[depth_index]->begin(), (*expanded_cube_indices_ptr)[depth_index]->end());
        i++;
    }

    auto last_expanded_depth_index_iterator = thrust::remove_copy_if(
            expanded_depth_per_cube_ptr->begin(),
            expanded_depth_per_cube_ptr->end(),
            thrust::make_counting_iterator(0), // Works as mask for values that should be copied (checked if identity is fulfilled)
            expanded_depth_per_cube_ptr->begin(),
    [dim_] __host__ __device__ (const int &val) { return val % int(pow(2,dim_)); });

    expanded_depth_per_cube_ptr->resize(last_expanded_depth_index_iterator - expanded_depth_per_cube_ptr->begin());

    if(monitor)
        print_range("Reduced expanded depth", expanded_depth_per_cube_ptr->begin(), expanded_depth_per_cube_ptr->end());

    compute_vertices(*expanded_cube_indices_ptr, *expanded_depth_per_cube_ptr);

    // Free memory
    delete expanded_depth_per_cube_ptr;
    //delete (*expanded_cube_indices_ptr)[0];
    thrust::for_each(expanded_cube_indices_ptr->begin(), expanded_cube_indices_ptr->end(), [] (dev_vec_int *elem) { delete elem; }); */
}
